#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
using namespace std;
#define N 10

void addVector(const int *a, const int *b, int *c, int number) {
	for (int i = 0; i < number; ++i) {
		c[i] = a[i] + b[i];
	}
}

__global__
void addVectorKernel(int *a, int *b, int *c, int number) {
	int tid = blockIdx.x;
	if (tid < number) {
		c[tid] = a[tid] + b[tid];
	}
}

int main()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 100, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	//addVector(a, b, c, 5);

	int size = 5 * sizeof(int);
	int *device_a, *device_b, *device_c;

	hipMalloc((void **)&device_a, size);
	hipMalloc((void **)&device_b, size);
	hipMalloc((void **)&device_c, size);

	hipMemcpy(device_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(device_b, b, size, hipMemcpyHostToDevice);
	//kernel_fuc<<<blockPerGrid, threadsPerBlock >> >();
	addVectorKernel << <N, 1 >> > (device_a, device_b, device_c, 5);


	hipMemcpy(c, device_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < 5; i++) {
		cout << c[i] << endl;
	}


	// ���»�ȡ�Կ�����
	hipDeviceProp_t deviceProp;
	int deviceCount;
	hipError_t hipError_t;
	hipError_t = hipGetDeviceCount(&deviceCount);
	for (int dev = 0; dev < deviceCount; dev++)
	{
		int driver_version{ 0 }, runtime_version{ 0 };
		hipDeviceProp_t device_prop;
		hipSetDevice(dev);
		/* hipGetDeviceProperties: ��ȡָ����GPU�豸���������Ϣ */
		hipGetDeviceProperties(&device_prop, dev);

		fprintf(stdout, "\n�豸 %d ����: %s\n", dev, device_prop.name);

		/* hipDriverGetVersion: ��ȡCUDA�����汾 */
		hipDriverGetVersion(&driver_version);
		fprintf(stdout, "CUDA�����汾�� %d.%d\n", driver_version / 1000, (driver_version % 1000) / 10);
		/* hipRuntimeGetVersion: ��ȡCUDA����ʱ�汾 */
		hipRuntimeGetVersion(&runtime_version);
		fprintf(stdout, "CUDA����ʱ�汾�� %d.%d\n", runtime_version / 1000, (runtime_version % 1000) / 10);

		fprintf(stdout, "�豸���������� %d.%d\n", device_prop.major, device_prop.minor);
		fprintf(stdout, "�豸�Ͽ��õ�ȫ���ڴ������� %f MB, %llu bytes\n",
			(float)device_prop.totalGlobalMem / (1024 * 1024), (unsigned long long)device_prop.totalGlobalMem);
		fprintf(stdout, "ÿһ���߳̿��Ͽ��õĹ����ڴ������� %f KB, %lu bytes\n",
			(float)device_prop.sharedMemPerBlock / 1024, device_prop.sharedMemPerBlock);
		fprintf(stdout, "ÿһ���߳̿��Ͽ��õ�32λ�Ĵ�������: %d\n", device_prop.regsPerBlock);
		fprintf(stdout, "һ���߳����������߳������� %d\n", device_prop.warpSize);
		fprintf(stdout, "���ڴ濽������������pitch��: %d bytes\n", device_prop.memPitch);
		fprintf(stdout, "ÿһ���߳̿���֧�ֵ�����߳�����: %d\n", device_prop.maxThreadsPerBlock);
		fprintf(stdout, "ÿһ���߳̿��ÿ��ά�ȵ�����С(x,y,z): (%d, %d, %d)\n",
			device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1], device_prop.maxThreadsDim[2]);
		fprintf(stdout, "ÿһ���̸߳��ÿ��ά�ȵ�����С(x,y,z): (%d, %d, %d)\n",
			device_prop.maxGridSize[0], device_prop.maxGridSize[1], device_prop.maxGridSize[2]);
		fprintf(stdout, "GPU���ʱ��Ƶ��: %.0f MHz (%0.2f GHz)\n",
			device_prop.clockRate*1e-3f, device_prop.clockRate*1e-6f);
		fprintf(stdout, "�豸�Ͽ��õĳ����ڴ�����: %lu bytes\n", device_prop.totalConstMem);
		fprintf(stdout, "�������Ҫ��: %lu bytes\n", device_prop.textureAlignment);
		fprintf(stdout, "�Ƿ�֧���豸�ص�����: %s\n", device_prop.deviceOverlap ? "Yes" : "No");
		fprintf(stdout, "�豸�϶ദ����������: %d\n", device_prop.multiProcessorCount);
		fprintf(stdout, "ִ�к˺���ʱ�Ƿ�������ʱ������: %s\n", device_prop.kernelExecTimeoutEnabled ? "Yes" : "No");
		fprintf(stdout, "�豸�Ƿ���һ������GPU: %s\n", device_prop.integrated ? "Yes" : "No");
		fprintf(stdout, "�豸�Ƿ�֧��ӳ�������ڴ�: %s\n", device_prop.canMapHostMemory ? "Yes" : "No");
		fprintf(stdout, "CUDA�豸����ģʽ: %d\n", device_prop.computeMode);
		fprintf(stdout, "һά����֧�ֵ�����С: %d\n", device_prop.maxTexture1D);
		fprintf(stdout, "��ά����֧�ֵ�����С(x,y): (%d, %d)\n", device_prop.maxTexture2D[0], device_prop.maxSurface2D[1]);
		fprintf(stdout, "��ά����֧�ֵ�����С(x,y,z): (%d, %d, %d)\n",
			device_prop.maxTexture3D[0], device_prop.maxSurface3D[1], device_prop.maxSurface3D[2]);
		fprintf(stdout, "�ڴ�ʱ��Ƶ�ʷ�ֵ: %.0f Mhz\n", device_prop.memoryClockRate * 1e-3f);
		fprintf(stdout, "ȫ���ڴ����߿��: %d bits\n", device_prop.memoryBusWidth);
		fprintf(stdout, "L2�����С: %d bytes\n", device_prop.l2CacheSize);
		fprintf(stdout, "ÿ���ദ����֧�ֵ�����߳�����: %d\n", device_prop.maxThreadsPerMultiProcessor);
		fprintf(stdout, "�豸�Ƿ�֧��ͬʱִ�ж���˺���: %s\n", device_prop.concurrentKernels ? "Yes" : "No");
		fprintf(stdout, "�첽��������: %d\n", device_prop.asyncEngineCount);
		fprintf(stdout, "�Ƿ�֧���豸����������һ��ͳһ�ĵ�ַ�ռ�: %s\n", device_prop.unifiedAddressing ? "Yes" : "No");
	}



	getchar();
	return 0;
}
